#pragma once
#include <opencv2/core.hpp>
#include "opencv2/core/utility.hpp"
#include "opencv2/highgui.hpp"
#include "opencv2/video.hpp"
#include "opencv2/cudaoptflow.hpp"
#include "opencv2/cudaarithm.hpp"

extern "C" {
#include <TH/TH.h>
}
#include <THC/THC.h>
#include "utils.h"

cv::cuda::GpuMat toGpuMat(THCState* state, THCudaTensor* tensor)
{
   int tensorDims = THCudaTensor_nDimension(state, tensor);
   int height = THCudaTensor_size(state, tensor, 0);
   int width = THCudaTensor_size(state, tensor, 1);
   int step = THCudaTensor_stride(state, tensor, 0);
   int type;

   if (tensorDims == 2) {
        // If such tensor is passed, assume that it is single-channel:
        type = CV_32F;
    } else if (tensorDims == 3) {
        // Otherwise depend on the 3rd dimension:
        int nChannels = THCudaTensor_size(state, tensor, 2);
        //printf("nChannels : %d\n", nChannels);
        switch(nChannels){
         case 1:
            type = CV_32FC1;
            //printf("CV_32FC1\n");
            break;
         case 2:
            type = CV_32FC2;
            //printf("CV_32FC2\n");
            break;
         case 3:
            type = CV_32FC3;
            //printf("CV_32FC3\n");
            break;
         case 4:
            type = CV_32FC4;
            //printf("CV_32FC4\n");
            break;
         default:
            THError("bad number of channels in toGpuMat, aborting");
        }
    }
    cv::cuda::GpuMat t(height, width, type, THCudaTensor_data(state, tensor), step*4);
    return t;
}

void computeOptFlow(THCState* state, THCudaTensor* image_a, THCudaTensor* image_b, THCudaTensor* flow, cv::Ptr<cv::cuda::DenseOpticalFlow> flowAlg, cv::cuda::Stream& s = cv::cuda::Stream::Null())
{
   cv::cuda::GpuMat img_a = toGpuMat(state, image_a);
   cv::cuda::GpuMat img_b = toGpuMat(state, image_b);
   cv::cuda::GpuMat flowout = toGpuMat(state, flow);

   flowAlg->calc(img_a,
               img_b,
               flowout,
               s);
}




// assume everything is pitched now and with correct sizes
static int cuof_computeOptFlow(lua_State *L)
{
   THCState *state = getCutorchState(L);
   THCudaTensor *input_a = (THCudaTensor *)luaT_checkudata(L, 1, "torch.CudaTensor");
   THCudaTensor *input_b = (THCudaTensor *)luaT_checkudata(L, 2, "torch.CudaTensor");
   THCudaTensor *output = (THCudaTensor *)luaT_checkudata(L, 3, "torch.CudaTensor");

   cv::Ptr<cv::cuda::BroxOpticalFlow> flowAlg = cv::cuda::BroxOpticalFlow::create();
   computeOptFlow(state, input_a, input_b, output, flowAlg);

   // check for errors
   hipError_t err = hipGetLastError();
   if (err != hipSuccess) {
     printf("error in computeOptFlow: %s\n", hipGetErrorString(err));
     THError("aborting");
   }
   return 1;
}

static int cuof_computeOptFlowBatch(lua_State *L)
{
   THCState *state = getCutorchState(L);
   THCudaTensor *inputs = (THCudaTensor *)luaT_checkudata(L, 1, "torch.CudaTensor");
   THCudaTensor *outputs = (THCudaTensor *)luaT_checkudata(L, 2, "torch.CudaTensor");

   THCudaTensor *input_a = THCudaTensor_new(state);
   THCudaTensor *input_b = THCudaTensor_new(state);
   THCudaTensor *output_n = THCudaTensor_new(state);

   cv::Ptr<cv::cuda::BroxOpticalFlow> flowAlg = cv::cuda::BroxOpticalFlow::create();

   int batchSize = THCudaTensor_size(state, inputs, 0);
   int elt;

   for(elt=0; elt<batchSize-1; elt++)
   {
      THCudaTensor_select(state, input_a, inputs, 0, elt);
      THCudaTensor_select(state, input_b, inputs, 0, elt+1);
      THCudaTensor_select(state, output_n, outputs, 0, elt);
      computeOptFlow(state, input_a, input_b, output_n, flowAlg);
   }

   // check for errors
  hipError_t err = hipGetLastError();
  if (err != hipSuccess) {
    printf("error in computeOptFlowBatch: %s\n", hipGetErrorString(err));
    THError("aborting");
  }

  return 1;
}

static int cuof_computeOptFlowBatchStreamed(lua_State *L)
{
   THCState *state = getCutorchState(L);
   THCudaTensor *inputs = (THCudaTensor *)luaT_checkudata(L, 1, "torch.CudaTensor");
   THCudaTensor *outputs = (THCudaTensor *)luaT_checkudata(L, 2, "torch.CudaTensor");

   THCudaTensor *input_a = THCudaTensor_new(state);
   THCudaTensor *input_b = THCudaTensor_new(state);
   THCudaTensor *output_n = THCudaTensor_new(state);

   cv::Ptr<cv::cuda::BroxOpticalFlow> flowAlg = cv::cuda::BroxOpticalFlow::create();

   int batchSize = THCudaTensor_size(state, inputs, 0);
   int elt;

   const int nStreams = 8;
   cv::cuda::Stream::Stream s[nStreams];

   for(elt=0; elt<batchSize-1; elt++)
   {
      THCudaTensor_select(state, input_a, inputs, 0, elt);
      THCudaTensor_select(state, input_b, inputs, 0, elt+1);
      THCudaTensor_select(state, output_n, outputs, 0, elt);
      computeOptFlow(state, input_a, input_b, output_n, flowAlg, s[elt % nStreams]);
   }

   // check for errors
  hipError_t err = hipGetLastError();
  if (err != hipSuccess) {
    printf("error in computeOptFlowBatch: %s\n", hipGetErrorString(err));
    THError("aborting");
  }

  return 1;
}

static const struct luaL_Reg cuof_ComputeOptFlow__ [] = {
  {"computeOptFlow", cuof_computeOptFlow},
  {"computeOptFlowBatch", cuof_computeOptFlowBatch},
  {"computeOptFlowBatchStreamed", cuof_computeOptFlowBatchStreamed},
  {NULL, NULL}
};

static void cuof_ComputeOptFlow_init(lua_State *L)
{
  luaT_pushmetatable(L, "torch.CudaTensor");
  luaT_registeratname(L, cuof_ComputeOptFlow__, "of");
  lua_pop(L,1);
}
